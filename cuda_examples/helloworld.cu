
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void myGpuFunction(int a, int b, int *c)
{
 *c=a+b;
}

int main(void)
{
int c;
int *dev_c;
hipMalloc((void**)&dev_c,sizeof(int));
myGpuFunction<<<1,1>>>(2,2,dev_c);
hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
printf("the Simple program to check make the GPU calls : Sum %d\n",c);
hipFree(dev_c);
return 0;
}
